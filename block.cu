#include "hip/hip_runtime.h"
// vim: noet ts=4 sts=4 sw=4

#include "block.h"
#include <iostream>

void cudaCheckError(hipError_t error, const char *file, const int line) {
	std::cerr << file << ":" << line << " CUDA error " << hipGetErrorString(error) << std::endl;
	std::terminate();
}

#define CUDA_CHECK_ERROR(__err) cudaCheckError(x, __FILE__, __LINE__)

biptr::biptr(size_t elems) : bytes(elems * sizeof(double)) {
	cudaCheckError(hipHostMalloc(&host, bytes));
	cudaCheckError(hipMalloc(&dev, bytes));
}

~biptr::biptr(size_t elems) {
	cudaCheckError(hipHostFree(host));
	cudaCheckError(hipFree(dev));
}

void biptr::dtoh() {
	cudaCheckError(hipMemcpy(host, device, bytes, hipMemcpyDeviceToHost));
}

void biptr::htod() {
	cudaCheckError(hipMemcpy(device, host, bytes, hipMemcpyHostToDevice));
}

void extractSlow(const SkewedBlock& block,
			double *Sxm, double *Sxp,
			double *Sym, double *Syp,
			double *Szm, double *Szp)
{
	int ni = block.ni;
	int nj = block.nj;
	int nk = block.nk;
	for (int j = 0; j < nj; j++) {
		for (int k = 0; k < nk; k++) {
			Sxm[j * nk + k] = block(0   , j, k);
			Sxp[j * nk + k] = block(ni-1, j, k);
		}
	}
	for (int i = 0; i < ni; i++) {
		for (int k = 0; k < nk; k++) {
			Sym[i * nk + k] = block(i, 0   , k);
			Syp[i * nk + k] = block(i, nj-1, k);
		}
	}
	for (int i = 0; i < ni; i++) {
		for (int j = 0; j < nj; j++) {
			Szm[i * nj + j] = block(i, j, 0   );
			Szp[i * nj + j] = block(i, j, nk-1);
		}
	}
}

void implodeSlow(SkewedBlock& block,
			const double *Sxm, const double *Sxp,
			const double *Sym, const double *Syp,
			const double *Szm, const double *Szp)
{
	int ni = block.ni;
	int nj = block.nj;
	int nk = block.nk;
	for (int j = 0; j < nj; j++) {
		for (int k = 0; k < nk; k++) {
			block(-1, j, k) = Sxm[j * nk + k];
			block(ni, j, k) = Sxp[j * nk + k];
		}
	}
	for (int i = 0; i < ni; i++) {
		for (int k = 0; k < nk; k++) {
			block(i, -1, k) = Sym[i * nk + k];
			block(i, nj, k) = Syp[i * nk + k];
		}
	}
	for (int i = 0; i < ni; i++) {
		for (int j = 0; j < nj; j++) {
			block(i, j, -1) = Szm[i * nj + j];
			block(i, j, nk) = Szp[i * nj + j];
		}
	}
}

void SkewedBlock::extract() {
	extractKernel<<<6, std::max(nj, nk)>>>(
			data.dev, ni, nj, nk,
			di, dj, dk,
			Sxm.dev, Sxp.dev,
			Sym.dev, Syp.dev,
			Szm.dev, Szp.dev);
}

void SkewedBlock::implode() {
	implodeKernel<<<6, std::max(nj, nk)>>>(
			data.dev, ni, nj, nk,
			di, dj, dk,
			Sxm.dev, Sxp.dev,
			Sym.dev, Syp.dev,
			Szm.dev, Szp.dev);
}

__global__ void extractKernel(
		const double *dataBase,
		const int ni, const int nj, const int nk,
		const int di, const int dj, const int dk,
		double *Sxm, double *Sxp,
		double *Sym, double *Syp,
		double *Szm, double *Szp)
{
	int side = blockIdx.x;
	int i, j, k;
	double *dst;
	if (side == 0) { // x-
		i = 0;
		dst = Sxm;
	} else if (side == 1) { // x+
		i = ni-1;
		dst = Sxp;
	} else if (side == 2) { // y-
		j = 0;
		dst = Sym;
	} else if (side == 3) { // y+
		j = nj-1;
		dst = Syp;
	} else if (side == 4) { // z-
		k = 0;
		dst = Szm;
	} else if (side == 5) { // z+
		k = nk-1;
		dst = Szp;
	}
	if (side == 0 || side == 1) {
		k = threadIdx.x;
		double *ptr = dataBase + i * di + k * dk;
		if (k < nk) {
			for (j = 0; j < nj; j++) {
				dst[j * nk + k] = ptr[j * dj];
			}
		}
	} else if (side == 2 || side == 3) {
		k = threadIdx.x;
		double *ptr = dataBase + j * dj + k * dk;
		if (k < nk) {
			for (i = 0; i < ni; i++) {
				dst[i * nk + k] = ptr[i * di];
			}
		}
	} else {
		j = threadIdx.x;
		double *ptr = dataBase + j * dj + k * dk;
		if (j < nj) {
			for (i = 0; i < ni; i++) {
				dst[i * nj + j] = ptr[i * di];
			}
		}
	}
}

__global__ void implodeKernel(
		const double *dataBase,
		const int ni, const int nj, const int nk,
		const int di, const int dj, const int dk,
		double *Sxm, double *Sxp,
		double *Sym, double *Syp,
		double *Szm, double *Szp)
{
	int side = blockIdx.y;
	int i, j, k;
	double *src;
	if (side == 0) { // x-
		i = -1;
		src = Sxm;
	} else if (side == 1) { // x+
		i = ni;
		src = Sxp;
	} else if (side == 2) { // y-
		j = -1;
		src = Sym;
	} else if (side == 3) { // y+
		j = nj;
		src = Syp;
	} else if (side == 4) { // z-
		k = -1;
		src = Szm;
	} else if (side == 5) { // z+
		k = nk;
		src = Szp;
	}
	if (side == 0 || side == 1) {
		k = threadIdx.x;
		double *ptr = dataBase + i * di + k * dk;
		if (k < nk) {
			for (j = 0; j < nj; j++) {
				ptr[j * dj] = src[j * nk + k];
			}
		}
	} else if (side == 2 || side == 3) {
		k = threadIdx.x;
		double *ptr = dataBase + j * dj + k * dk;
		if (k < nk) {
			for (i = 0; i < ni; i++) {
				ptr[i * di] = src[i * nk + k];
			}
		}
	} else {
		j = threadIdx.x;
		double *ptr = dataBase + j * dj + k * dk;
		if (j < nj) {
			for (i = 0; i < ni; i++) {
				ptr[i * di] = src[i * nj + j];
			}
		}
	}
}
